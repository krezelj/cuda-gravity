#include "gravity_box.cuh"
#include <cstdio>

GravityBox::GravityBox(BodyArray* bodies, float delta_t) : bodies(bodies)
{
	this->delta_t = delta_t;
	this->N = &(bodies->N);

	ax = new float[*N];
	ay = new float[*N];
	collision = new int[*N];

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	
	d_acceleration = 0;
	cudaStatus = hipMalloc((void**)&d_acceleration, *N * 2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	d_g_mass = 0;
	cudaStatus = hipMalloc((void**)&d_g_mass, *N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	d_position_x = 0;
	cudaStatus = hipMalloc((void**)&d_position_x, *N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	d_position_y = 0;
	cudaStatus = hipMalloc((void**)&d_position_y, *N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	d_velocity_x = 0;
	cudaStatus = hipMalloc((void**)&d_velocity_x, *N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	d_velocity_y = 0;
	cudaStatus = hipMalloc((void**)&d_velocity_y, *N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	d_collision = 0;
	cudaStatus = hipMalloc((void**)&d_collision, *N * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	// copy data to gpu
	// NOTE: While using GPU mode only in-gpu properties are updates (except x and y)
	// g_mass
	cudaStatus = hipMemcpy(d_g_mass, bodies->g_mass, *N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_g_mass);
	}

	// x
	cudaStatus = hipMemcpy(d_position_x, bodies->x, *N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_position_x);
	}

	// y
	cudaStatus = hipMemcpy(d_position_y, bodies->y, *N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_position_y);
	}

	// vx
	cudaStatus = hipMemcpy(d_velocity_x, bodies->vx, *N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_velocity_x);
	}


	// vy
	cudaStatus = hipMemcpy(d_velocity_y, bodies->vy, *N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_velocity_y);
	}
}

GravityBox::~GravityBox()
{
	hipFree(d_acceleration);
	hipFree(d_g_mass);
	hipFree(d_position_x);
	hipFree(d_position_y);
	hipFree(d_velocity_x);
	hipFree(d_velocity_y);
	hipFree(d_collision);

	delete[] ax;
	delete[] ay;
	delete[] collision;

	hipDeviceReset();
}

void GravityBox::UpdateSimulation(GB_MODE mode, int n_steps)
{
	if (mode == GB_USE_CPU)
	{
		for (int step = 0; step < n_steps; step++)
		{
			UpdateAccelerationsCPU();
			UpdateBodiesCPU();
		}
	}
	else 
	{
		UpdateSimulationGPU(n_steps);
	}	
}